#include "hip/hip_runtime.h"
// -*- c++-mode -*-
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <cassert>
#include <gflags/gflags.h>

DEFINE_int32(kernel_grid_dim, 32, "[CUDA]: kernel's grid (square) size.");
DEFINE_int32(kernel_block_dim, 16, "[CUDA]: kernel's block (square) size.");
//DEFINE_int32(kernel_block_mult, 4, "[CUDA]: multiplicity of the sub-image each block works on.");
DEFINE_bool(validate, false, "Check the correctness of the result against reference CPU implementation.");
//DEFINE_bool(perf_test, false, "Whether to perform test the implementation.");
DEFINE_bool(random_init, false, "Whether to create randomly initialized data.");

#define CHECK_ERROR(...) {                                  \
        hipGetLastError();                                 \
        __VA_ARGS__;                                        \
        hipError_t err = hipPeekAtLastError();            \
        if ( hipSuccess != err ) {                         \
            printf("[CUDA ERROR] => %s\n\tmsg: %s\n",		\
                   #__VA_ARGS__, hipGetErrorString(err));	\
        }                                                   \
    }

template <typename T>
struct Image3D {
    size_t nrows, ncols, nz;
    size_t nbytes;
    T *data;

    T get(int z, int x, int y) {
        return data[x + ncols * (y + nrows * z)];
    }

    void set(int z, int x, int y, T val) {
        data[x + ncols * (y + nrows * z)] = val;
    }

    Image3D(size_t nrows, size_t ncols, size_t nz)
        : nrows(nrows), ncols(ncols), nz(nz) {
        nbytes = nrows * ncols * nz * sizeof(T);
        data = new T[nrows * ncols * nz];
    }

    ~Image3D() {
        delete [] data;
    }

    void print() {
        for (int z = 0; z < nz; ++z) {
            for (int y = 0; y < nrows; ++y) {
                for (int x = 0; x < ncols; ++x) {
                    printf("%3d ", get(z, x, y));
                }
                std::cout << std::endl;
            }
            std::cout << "<--- z: " << z << "--->" << std::endl;
        }
        std::cout << "-----------------------------------" << std::endl;
    }
};

template <typename T>
__global__ void maxPoll2D_naive(const T *img_src, 
                                size_t nrows, size_t ncols, size_t nz,
                                T *img_dst, size_t K) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int tile_dim_x = blockDim.x * gridDim.x;
    int tile_dim_y = blockDim.y * gridDim.y;

    const T MIN_VAL = (T) INT_MIN;

    size_t nx = ncols - K + 1;
    size_t ny = nrows - K + 1;

    for (int z = 0; z < nz; ++z) {
        T *zimg_src = img_src + z * ncols * nrows;
        T *zimg_dst = img_dst + z * ny * nx;

        for (int y = idy; y < ny; y += tile_dim_y) {
            for (int x = idx; x < nx; x += tile_dim_x) {
                T max_val = MIN_VAL;
                for (int iy = y; iy < y + K; ++iy) {
                    T *pwnd = zimg_src + iy * ncols + x;
                    for (int i = 0; i < K; ++i)
                        max_val = max(max_val, *pwnd++);
                }
                zimg_dst[x + nx * y] = max_val;
            }
        }
    }
}

/**
 * With a block dimension, too
 */
template <typename T, int BLK_M>
__global__ void maxPoll2D(const T *img_src, 
                          int nrows, int ncols, int nz,
                          T *img_dst, int K) {
    const T MIN_VAL = (T) INT_MIN;

    int dst_nrows = nrows - K + 1;
    int dst_ncols = ncols - K + 1;
    int dst_nbx = BLK_M * blockDim.x;
    int dst_nby = BLK_M * blockDim.y;
    int dst_stride_x = dst_nbx * gridDim.x;
    int dst_stride_y = dst_nby * gridDim.y;
    int src_nbx = dst_nbx + K - 1;
    int src_nby = dst_nby + K - 1;

    extern __shared__ T shmem_img_src[]; // at least src_nbx * src_nby * sizeof(T)

    for (int z = blockIdx.z; z < nz; z += gridDim.z) {
        const T *zimg_src = img_src + z * ncols * nrows;
        T *zimg_dst = img_dst + z * dst_ncols * dst_nrows;

        for (int by = blockIdx.y * dst_nby;
             by < dst_nrows; by += dst_stride_y) {
            for (int bx = blockIdx.x * dst_nbx;
                 bx < dst_ncols; bx += dst_stride_x) {

                // Copy data into shared memory
                int src_bndx = min(src_nbx, ncols - bx);
                int src_bndy = min(src_nby, nrows - by);
                for (int ty = threadIdx.y;
                     ty < src_bndy; ty += blockDim.y) {
                    for (int tx = threadIdx.x;
                         tx < src_bndx; tx += blockDim.x) {
                        shmem_img_src[ tx + src_nbx * ty ] =
                            zimg_src[ tx + bx + ncols * (ty + by) ];
                    }
                }
                __syncthreads();

                // Compute max polling
                int dst_bndx = min(dst_nbx, dst_ncols - bx);
                int dst_bndy = min(dst_nby, dst_nrows - by);
                for (int ty = threadIdx.y;
                     ty < dst_bndy; ty += blockDim.y) {
                    for (int tx = threadIdx.x;
                         tx < dst_bndx; tx += blockDim.x) {
                        T max_val = MIN_VAL;
                        for (int i = 0; i < K; ++i) {
                            for (int j = 0; j < K; ++j) {
                                max_val = max(max_val,
                                              shmem_img_src[tx + j + src_nbx * (ty + i)]);
                            }
                        }
                        zimg_dst[ tx + bx + dst_ncols * (ty + by) ] = max_val;
                    }
                }
                __syncthreads(); // !! must sync before proceeding to next loop
            }
        }
    }
}

struct KernelProfile {
    dim3 grid_dim;
    dim3 block_dim;

    void print(std::ostream &ostrm = std::cerr) {
        ostrm << "[CUDA]  grid size: "
              << "x = " << grid_dim.x << " "
              << "y = " << grid_dim.y << " "
              << "z = " << grid_dim.z << std::endl;
        ostrm << "[CUDA] block size: "
              << "x = " << block_dim.x << " "
              << "y = " << block_dim.y << " "
              << "z = " << block_dim.z << std::endl;
    }
};

template <typename T, int BLK_M>
Image3D<T> maxPollGPU(const Image3D<T> &img_orig, int K,
                      const KernelProfile &kernel_profile) {
    // Size of the max polling kernel
    int nrows = img_orig.nrows;
    int ncols = img_orig.ncols;
    int nz = img_orig.nz;
    Image3D<int> img_poll(nrows - K + 1, ncols - K + 1, nz);

    // Allocate GPU memory
    int *d_img_src, *d_img_dst;

    size_t IMG_SRC_NBYTES = img_orig.nbytes;
    size_t IMG_DST_NBYTES = img_poll.nbytes;
    CHECK_ERROR( hipMalloc((void **) &d_img_src, IMG_SRC_NBYTES) );
    CHECK_ERROR( hipMalloc((void **) &d_img_dst, IMG_DST_NBYTES) );

    int *h_img_src = img_orig.data;
    int *h_img_dst = img_poll.data;

    //dim3 cuGridDim(32, 32, nz), cuBlockDim(16, 16, 1);
    dim3 cuGridDim = kernel_profile.grid_dim;
    dim3 cuBlockDim = kernel_profile.block_dim;
  
    size_t SHMEM_NBYTES = 256 + sizeof(int) *
        (BLK_M * cuBlockDim.x + K - 1) *
        (BLK_M * cuBlockDim.y + K - 1);

    printf("Computing max polling for image size: %d %d %d, K = %d\n", nrows, ncols, nz, K);
    CHECK_ERROR( hipMemcpy(d_img_src, h_img_src, IMG_SRC_NBYTES, hipMemcpyHostToDevice) );
    CHECK_ERROR( maxPoll2D<int, BLK_M>
                 <<<cuGridDim, cuBlockDim, SHMEM_NBYTES>>>(d_img_src, nrows, ncols, nz, d_img_dst, K) );
    //CHECK_ERROR( maxPoll2D_naive<int><<<cuGridDim, cuBlockDim>>>(d_img_src, nrows, ncols, nz, d_img_dst, K) );
    CHECK_ERROR( hipMemcpy(h_img_dst, d_img_dst, IMG_DST_NBYTES, hipMemcpyDeviceToHost) );

    hipFree(d_img_src);
    hipFree(d_img_dst);
    return img_poll;
}

int main(int argc, char **argv) {
    // // img_orig.print();
    gflags::ParseCommandLineFlags(&argc, &argv, true);
  
    int nrows, ncols, nz, K;
    bool is_parse_stdin = false;
    if ( 5 == argc ) {
        std::cerr << "Parsing input sizes from stdin" << std::endl;
        nrows = atoi(argv[1]);
        ncols = atoi(argv[2]);
        nz = atoi(argv[3]);
        K = atoi(argv[4]);
    } else {
        std::cerr << "Parsing from stdin" << std::endl;
        std::cin >> nrows >> ncols >> nz;
        is_parse_stdin = true;
    }

    Image3D<int> img_orig(nrows, ncols, nz);
    if ( is_parse_stdin ) {
        for (int z = 0; z < nz; ++z) {
            for (int y = 0; y < nrows; ++y) {
                for (int x = 0; x < ncols; ++x) {
                    int val; std::cin >> val;
                    img_orig.set(z, x, y, val);
                }
            }
        }
        std::cin >> K;
    }

    // Randomly initialize the input
    if ( FLAGS_random_init ) {
        std::cerr << "Randomly initializing input, might take a while ..." << std::endl;
        for (int z = 0; z < nz; ++z) {
            for (int y = 0; y < nrows; ++y)
                for (int x = 0; x < ncols; ++x)
                    img_orig.set(z, x, y, rand() % 256);
        }
    }

    dim3 cuGridDim(FLAGS_kernel_grid_dim, FLAGS_kernel_grid_dim, nz);
    dim3 cuBlockDim(FLAGS_kernel_block_dim, FLAGS_kernel_block_dim, 1);
    KernelProfile kernel_profile;
    kernel_profile.grid_dim = cuGridDim;
    kernel_profile.block_dim = cuBlockDim;
    kernel_profile.print();
  
    Image3D<int> img_poll = maxPollGPU<int, 3>(img_orig, K, kernel_profile);
    //cout << img_poll.nrows << " " << img_poll.ncols << endl;  

    // Check the result
    if ( FLAGS_validate ) {
        std::cerr << "Validating result, might take a while ... ";
        int num_err = 0;
        for (int z = 0; z < nz; ++z)
            for (int y = 0; y < nrows - K + 1; ++y)
                for (int x = 0; x < ncols - K + 1; ++x) {
                    int ref_max = INT_MIN;
                    for (int i = 0; i < K; ++i)
                        for (int j = 0; j < K; ++j)
                            ref_max = max(ref_max, img_orig.get(z, x + j, y + i));

                    int val = img_poll.get(z, x, y);
                    if ( val != ref_max ) {
                        // printf("z = %d, x = %d, y = %d, val = %d, ref = %d\n",
                        // 	 z, x, y, val, ref_max);
                        ++num_err;
                    }
                }
        if ( 0 == num_err )
            std::cerr << "PASSED !" << std::endl;
        else
            std::cerr << "FAILED with error counts: " << num_err << std::endl;
    }
    //assert( 0 == num_err );
    //img_poll.print();
}
